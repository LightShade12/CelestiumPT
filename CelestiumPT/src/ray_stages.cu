#include "hip/hip_runtime.h"
#include "ray_stages.cuh"
#include "ray.cuh"
#include "storage.cuh"
#include "scene_geometry.cuh"
#include "shape_intersection.cuh"
#include "triangle.cuh"
#include "maths/constants.cuh"

__device__ ShapeIntersection MissStage(const IntegratorGlobals& globals, const Ray& ray, const ShapeIntersection& in_payload) {
	ShapeIntersection out_payload;
	out_payload.GAS_debug = in_payload.GAS_debug;
	out_payload.hit_count = in_payload.hit_count;
	return out_payload;
}

__device__ ShapeIntersection ClosestHitStage(const IntegratorGlobals& globals, const Ray& ray, const ShapeIntersection& in_payload)
{
	const Triangle& triangle = globals.SceneDescriptor.DeviceGeometryAggregate->DeviceTrianglesBuffer[in_payload.triangle_idx];

	ShapeIntersection out_payload;

	out_payload.bary = in_payload.bary;
	out_payload.triangle_idx = in_payload.triangle_idx;
	out_payload.hit_distance = in_payload.hit_distance;
	out_payload.GAS_debug = in_payload.GAS_debug;
	out_payload.m_invModelMatrix = in_payload.m_invModelMatrix;
	out_payload.arealight = in_payload.arealight;
	out_payload.object_idx = in_payload.object_idx;
	out_payload.hit_count = in_payload.hit_count;

	Mat4 model_matrix = in_payload.m_invModelMatrix.inverse();
	out_payload.w_pos = ray.getOrigin() + (ray.getDirection() * in_payload.hit_distance);
	out_payload.l_pos = make_float3(in_payload.m_invModelMatrix * make_float4(out_payload.w_pos, 1));

	out_payload.w_geo_norm = triangle.face_normal;

	out_payload.uv =
		(triangle.vertex0.UV * out_payload.bary.x) +
		(triangle.vertex1.UV * out_payload.bary.y) +
		(triangle.vertex2.UV * out_payload.bary.z);

	out_payload.w_shading_norm = normalize(
		(triangle.vertex0.normal * out_payload.bary.x) +
		(triangle.vertex1.normal * out_payload.bary.y) +
		(triangle.vertex2.normal * out_payload.bary.z)
	);
	out_payload.l_shading_norm = out_payload.w_shading_norm;

	if (dot(
		normalize(make_float3(model_matrix * make_float4(out_payload.w_geo_norm, 0))),
		-1 * ray.getDirection()) < 0.f)
	{
		out_payload.front_face = false;
		out_payload.w_geo_norm = normalize(make_float3(model_matrix * make_float4(-1.f * out_payload.w_geo_norm, 0)));
		out_payload.w_shading_norm = normalize(make_float3(model_matrix * make_float4(-1.f * out_payload.w_shading_norm, 0)));
	}
	else {
		out_payload.w_geo_norm = normalize(make_float3(model_matrix * make_float4(out_payload.w_geo_norm, 0)));
		out_payload.w_shading_norm = normalize(make_float3(model_matrix * make_float4(out_payload.w_shading_norm, 0)));
		out_payload.front_face = true;
	}

	return out_payload;
}

__device__ void IntersectionStage(const Ray& ray, const Triangle& triangle, int triangle_idx, CompactShapeIntersection* payload)
{
	payload->hit_distance = FLT_MAX;
	payload->triangle_idx = -1;

	float3 v0v1 = triangle.vertex1.position - triangle.vertex0.position;
	float3 v0v2 = triangle.vertex2.position - triangle.vertex0.position;

	float3 pvec = cross(ray.getDirection(), v0v2);

	float det = dot(v0v1, pvec);
	if (det > -TRIANGLE_EPSILON && det < TRIANGLE_EPSILON)
		return; //payload; // This ray is parallel to this triangle

	float invDet = 1.0f / det;
	float3 tvec = ray.getOrigin() - triangle.vertex0.position;
	float u = invDet * dot(tvec, pvec);
	if (u < 0.0f || u > 1.0f)
		return;// payload;

	float3 qvec = cross(tvec, v0v1);
	float v = invDet * dot(ray.getDirection(), qvec);
	if (v < 0.0f || u + v > 1.0f)
		return;// payload;

	float t = invDet * dot(v0v2, qvec);
	if (t > TRIANGLE_EPSILON) { // ray intersection
		payload->hit_distance = t;
		payload->triangle_idx = triangle_idx;
		payload->bary = { 1.0f - u - v, u, v };
		return; //payload;
	}

	return;// payload;
}
__device__ bool AnyHitStage(const Ray& t_ray, const SceneGeometry& t_scene_geometry, const CompactShapeIntersection& t_payload)
{
	return true;
}
;