#include "hip/hip_runtime.h"
#include "Renderer.hpp"
#include "ErrorCheck.cuh"
#include "Storage.cuh"
#include "Integrator.cuh"

#include "SceneGeometry.cuh"
#include "DeviceScene.cuh"
#include "DeviceCamera.cuh"
//#include "Triangle.cuh"

#include <cuda_gl_interop.h>
#include <iostream>
/*
* Mostly adapter and inter conversion work
*/

class FrameBuffer {
public:

	FrameBuffer() = default;

	void initialize(int width, int height)
	{
		//GL texture configure
		glGenTextures(1, &m_RenderTargetTextureName);
		glBindTexture(GL_TEXTURE_2D, m_RenderTargetTextureName);

		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
		//TODO: make a switchable frame filtering mode
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA, GL_FLOAT, NULL);

		glBindTexture(GL_TEXTURE_2D, 0);

		hipGraphicsGLRegisterImage(&m_RenderTargetTextureCudaResource, m_RenderTargetTextureName, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
	};

	~FrameBuffer() {
		glDeleteTextures(1, &m_RenderTargetTextureName);
	}

	void resizeResolution(int width, int height) {
		if (m_RenderTargetTextureName == NULL) {
			initialize(width, height); return;
		}
		// unregister
		hipGraphicsUnregisterResource(m_RenderTargetTextureCudaResource);
		// resize
		glBindTexture(GL_TEXTURE_2D, m_RenderTargetTextureName);
		{
			glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA, GL_FLOAT, NULL);
		}
		glBindTexture(GL_TEXTURE_2D, 0);
		// register back
		hipGraphicsGLRegisterImage(&m_RenderTargetTextureCudaResource, m_RenderTargetTextureName, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
	};

	void beginRender(hipSurfaceObject_t* surf_obj) {
		hipGraphicsMapResources(1, &m_RenderTargetTextureCudaResource);

		hipArray_t render_target_texture_sub_resource_array;
		hipGraphicsSubResourceGetMappedArray(&render_target_texture_sub_resource_array, m_RenderTargetTextureCudaResource, 0, 0);
		hipResourceDesc render_target_texture_resource_descriptor;
		{
			render_target_texture_resource_descriptor.resType = hipResourceTypeArray;
			render_target_texture_resource_descriptor.res.array.array = render_target_texture_sub_resource_array;
		}

		//prepare globals--------------------
		hipCreateSurfaceObject(surf_obj, &render_target_texture_resource_descriptor);
	}

	void endRender(hipSurfaceObject_t* surf_obj)
	{
		hipDestroySurfaceObject(*surf_obj);
		hipGraphicsUnmapResources(1, &m_RenderTargetTextureCudaResource);
		hipStreamSynchronize(0);
	}

	GLuint m_RenderTargetTextureName = NULL;
	hipGraphicsResource_t m_RenderTargetTextureCudaResource;
};

struct CudaAPI
{
	dim3 m_BlockGridDimensions;
	dim3 m_ThreadBlockDimensions;
};

struct CelestiumPT_API
{
	DeviceScene DeviceScene;
	IntegratorGlobals m_IntegratorGlobals;
	FrameBuffer CompositeRenderBuffer;
	FrameBuffer NormalsRenderBuffer;
	FrameBuffer PositionsRenderBuffer;
	FrameBuffer DepthRenderBuffer;
	FrameBuffer LocalPositionsRenderBuffer;
	FrameBuffer GASDebugRenderBuffer;
	FrameBuffer UVsDebugRenderBuffer;
	FrameBuffer BarycentricsDebugRenderBuffer;
	FrameBuffer ObjectIDDebugRenderBuffer;
	FrameBuffer ObjectIDRenderBuffer;
	FrameBuffer VelocityRenderBuffer;

	FrameBuffer HistoryColorRenderFrontBuffer;//read
	FrameBuffer HistoryColorRenderBackBuffer;//write
	FrameBuffer HistoryDepthRenderBuffer;

	thrust::device_vector<float3>AccumulationFrameBuffer;
};

static GLenum fboStatus = GL_FRAMEBUFFER_INCOMPLETE_ATTACHMENT;
static bool blit_fbo_init = false;

Renderer::Renderer()
{
	m_CudaResourceAPI = new CudaAPI();
	m_CelestiumPTResourceAPI = new CelestiumPT_API();
	//TODO: redundant
	m_CudaResourceAPI->m_BlockGridDimensions = dim3(m_NativeRenderResolutionWidth / m_ThreadBlock_x + 1, m_NativeRenderResolutionHeight / m_ThreadBlock_y + 1);
	m_CudaResourceAPI->m_ThreadBlockDimensions = dim3(m_ThreadBlock_x, m_ThreadBlock_y);

	hipMallocManaged(&m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.device_geometry_aggregate, sizeof(SceneGeometry));

	m_CelestiumPTResourceAPI->DeviceScene = DeviceScene(m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.device_geometry_aggregate);
	m_CurrentScene = HostScene(&(m_CelestiumPTResourceAPI->DeviceScene));
}

void Renderer::resizeResolution(int width, int height)
{
	if (width == m_NativeRenderResolutionWidth && height == m_NativeRenderResolutionHeight)return;
	m_NativeRenderResolutionHeight = height;
	m_NativeRenderResolutionWidth = width;

	m_CelestiumPTResourceAPI->CompositeRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->NormalsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->PositionsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->DepthRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryColorRenderFrontBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryColorRenderBackBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->GASDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->UVsDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->BarycentricsDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->ObjectIDRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->VelocityRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	m_CudaResourceAPI->m_BlockGridDimensions = dim3(m_NativeRenderResolutionWidth / m_ThreadBlock_x + 1, m_NativeRenderResolutionHeight / m_ThreadBlock_y + 1);
	m_CudaResourceAPI->m_ThreadBlockDimensions = dim3(m_ThreadBlock_x, m_ThreadBlock_y);

	m_CelestiumPTResourceAPI->AccumulationFrameBuffer.resize(m_NativeRenderResolutionHeight * m_NativeRenderResolutionWidth);

	//TODO: workaround for error in constructor execution
	if (!blit_fbo_init) {
		glGenFramebuffers(1, &m_blit_mediator_FBO_name);
		// enable this frame buffer as the current frame buffer
		glBindFramebuffer(GL_FRAMEBUFFER, m_blit_mediator_FBO_name);
		// attach the textures to the frame buffer
		glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D,
			m_CelestiumPTResourceAPI->HistoryColorRenderBackBuffer.m_RenderTargetTextureName, 0);
		glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT1, GL_TEXTURE_2D,
			m_CelestiumPTResourceAPI->HistoryColorRenderFrontBuffer.m_RenderTargetTextureName, 0);
		glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT2, GL_TEXTURE_2D,
			m_CelestiumPTResourceAPI->DepthRenderBuffer.m_RenderTargetTextureName, 0);
		glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT3, GL_TEXTURE_2D,
			m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer.m_RenderTargetTextureName, 0);
		fboStatus = glCheckFramebufferStatus(GL_FRAMEBUFFER);
		if (fboStatus != GL_FRAMEBUFFER_COMPLETE) {
			printf(">[FRAMEBUFFER INCOMPLETE: 0x%x ]\n", fboStatus);
			//exit(1);
		}
		glBindFramebuffer(GL_FRAMEBUFFER, 0);
		blit_fbo_init = true;
	}
}

static uint32_t g_frameIndex = 1;

void Renderer::renderFrame()
{
	//pre render-------------------------------------------------------------

	m_CelestiumPTResourceAPI->CompositeRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.composite_render_surface_object));
	m_CelestiumPTResourceAPI->NormalsRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.normals_render_surface_object));
	m_CelestiumPTResourceAPI->PositionsRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.positions_render_surface_object));
	m_CelestiumPTResourceAPI->DepthRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.depth_render_surface_object));
	m_CelestiumPTResourceAPI->HistoryColorRenderFrontBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_color_render_front_surface_object));
	m_CelestiumPTResourceAPI->HistoryColorRenderBackBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_color_render_back_surface_object));
	m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_depth_render_surface_object));
	m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.local_positions_render_surface_object));
	m_CelestiumPTResourceAPI->GASDebugRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.GAS_debug_render_surface_object));
	m_CelestiumPTResourceAPI->UVsDebugRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.UV_debug_render_surface_object));
	m_CelestiumPTResourceAPI->BarycentricsDebugRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.bary_debug_render_surface_object));
	m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.objectID_debug_render_surface_object));
	m_CelestiumPTResourceAPI->ObjectIDRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.objectID_render_surface_object));
	m_CelestiumPTResourceAPI->VelocityRenderBuffer.beginRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.velocity_render_surface_object));

	//prepare globals--------------------
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.frameidx = g_frameIndex;
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.resolution =
		make_int2(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.accumulation_framebuffer =
		thrust::raw_pointer_cast(m_CelestiumPTResourceAPI->AccumulationFrameBuffer.data());

	IntegratorPipeline::invokeRenderKernel(m_CelestiumPTResourceAPI->m_IntegratorGlobals,
		m_CudaResourceAPI->m_BlockGridDimensions, m_CudaResourceAPI->m_ThreadBlockDimensions);
	g_frameIndex++;

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	//----

	//post render cuda---------------------------------------------------------------------------------
	m_CelestiumPTResourceAPI->CompositeRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.composite_render_surface_object));
	m_CelestiumPTResourceAPI->UVsDebugRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.UV_debug_render_surface_object));
	m_CelestiumPTResourceAPI->BarycentricsDebugRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.bary_debug_render_surface_object));
	m_CelestiumPTResourceAPI->NormalsRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.normals_render_surface_object));
	m_CelestiumPTResourceAPI->DepthRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.depth_render_surface_object));
	m_CelestiumPTResourceAPI->HistoryColorRenderFrontBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_color_render_front_surface_object));
	m_CelestiumPTResourceAPI->HistoryColorRenderBackBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_color_render_back_surface_object));
	m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_depth_render_surface_object));
	m_CelestiumPTResourceAPI->PositionsRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.positions_render_surface_object));
	m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.local_positions_render_surface_object));
	m_CelestiumPTResourceAPI->GASDebugRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.GAS_debug_render_surface_object));
	m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.objectID_debug_render_surface_object));
	m_CelestiumPTResourceAPI->ObjectIDRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.objectID_render_surface_object));
	m_CelestiumPTResourceAPI->VelocityRenderBuffer.endRender(
		&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.velocity_render_surface_object));

	//do blit
	if (fboStatus == GL_FRAMEBUFFER_COMPLETE) {
		glBindFramebuffer(GL_FRAMEBUFFER, m_blit_mediator_FBO_name);
		//color
		glReadBuffer(GL_COLOR_ATTACHMENT0);
		glDrawBuffers(1, &m_blit_target0_attachment);
		glBlitFramebuffer(0, 0, m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight,
			0, 0, m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight,
			GL_COLOR_BUFFER_BIT, GL_NEAREST);
		//depth
		glReadBuffer(GL_COLOR_ATTACHMENT2);
		glDrawBuffers(1, &m_blit_target1_attachment);
		glBlitFramebuffer(0, 0, m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight,
			0, 0, m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight,
			GL_COLOR_BUFFER_BIT, GL_NEAREST);

		glBindFramebuffer(GL_FRAMEBUFFER, 0);
	}
}

void Renderer::clearAccumulation()
{
	if (!m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.accumulate)return;

	thrust::fill(m_CelestiumPTResourceAPI->AccumulationFrameBuffer.begin(),
		m_CelestiumPTResourceAPI->AccumulationFrameBuffer.end(), make_float3(0, 0, 0));

	g_frameIndex = 1;
}

GLuint Renderer::getCompositeRenderTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->CompositeRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getNormalsTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->NormalsRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getPositionsTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->PositionsRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getLocalPositionsTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getVelocityTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->VelocityRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getGASDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->GASDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getDepthTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->DepthRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getHistoryDepthTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getUVsDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->UVsDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getObjectIDDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getBarycentricsDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->BarycentricsDebugRenderBuffer.m_RenderTargetTextureName;
}

void Renderer::setCamera(int idx)
{
	if (idx < 0)idx = 0;
	if (idx >= m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras.size())idx = m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras.size() - 1;
	DeviceCamera* dcam = thrust::raw_pointer_cast(&m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras[idx]);
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.active_camera = dcam;
	m_CurrentCamera = HostCamera(m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.active_camera);
}

IntegratorSettings* Renderer::getIntegratorSettings()
{
	return &(m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG);
}

Renderer::~Renderer()
{
	glDeleteFramebuffers(1, &m_blit_mediator_FBO_name);
	hipFree(m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.device_geometry_aggregate);//TODO: non critical ownership issues with devicescene
	delete m_CudaResourceAPI;
	m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras.clear();
	delete m_CelestiumPTResourceAPI;
}