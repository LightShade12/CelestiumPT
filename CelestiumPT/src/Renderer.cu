#include "hip/hip_runtime.h"
#include "renderer.hpp"
#include "error_check.cuh"
#include "storage.cuh"
#include "integrator.cuh"
#include "svgf/svgf_passes.cuh"
#include "svgf/svgf_temporal_reprojection.cuh"
#include "render_passes.cuh"
#include "temporal_pass.cuh"

#include "scene_geometry.cuh"
#include "device_scene.cuh"
#include "device_camera.cuh"
//#include "Triangle.cuh"

#include <cuda_gl_interop.h>
#include <iostream>
/*
* Mostly adapter and inter conversion work
*/

class FrameBuffer {
public:

	FrameBuffer() = default;

	void initialize(int width, int height)
	{
		//GL texture configure
		glGenTextures(1, &m_RenderTargetTextureName);
		glBindTexture(GL_TEXTURE_2D, m_RenderTargetTextureName);

		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
		//TODO: make a switchable frame filtering mode
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA, GL_FLOAT, NULL);

		glBindTexture(GL_TEXTURE_2D, 0);

		hipGraphicsGLRegisterImage(&m_RenderTargetTextureCudaResource, m_RenderTargetTextureName, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
	};

	~FrameBuffer() {
		glDeleteTextures(1, &m_RenderTargetTextureName);
	}

	void resizeResolution(int width, int height) {
		if (m_RenderTargetTextureName == NULL) {
			initialize(width, height); return;
		}
		// unregister
		hipGraphicsUnregisterResource(m_RenderTargetTextureCudaResource);
		// resize
		glBindTexture(GL_TEXTURE_2D, m_RenderTargetTextureName);
		{
			glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA, GL_FLOAT, NULL);
		}
		glBindTexture(GL_TEXTURE_2D, 0);
		// register back
		hipGraphicsGLRegisterImage(&m_RenderTargetTextureCudaResource, m_RenderTargetTextureName, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
	};

	void beginRender(hipSurfaceObject_t* surf_obj) {
		hipGraphicsMapResources(1, &m_RenderTargetTextureCudaResource);

		hipArray_t render_target_texture_sub_resource_array;
		hipGraphicsSubResourceGetMappedArray(&render_target_texture_sub_resource_array, m_RenderTargetTextureCudaResource,
			0, 0);
		hipResourceDesc render_target_texture_resource_descriptor;
		{
			render_target_texture_resource_descriptor.resType = hipResourceTypeArray;
			render_target_texture_resource_descriptor.res.array.array = render_target_texture_sub_resource_array;
		}

		//prepare globals--------------------
		hipCreateSurfaceObject(surf_obj, &render_target_texture_resource_descriptor);
	}

	void endRender(hipSurfaceObject_t* surf_obj)
	{
		hipDestroySurfaceObject(*surf_obj);
		hipGraphicsUnmapResources(1, &m_RenderTargetTextureCudaResource);
		hipStreamSynchronize(0);
	}

	GLuint m_RenderTargetTextureName = NULL;
	hipGraphicsResource_t m_RenderTargetTextureCudaResource;
};

struct CudaAPI
{
	dim3 m_BlockGridDimensions;
	dim3 m_ThreadBlockDimensions;
};

struct CelestiumPT_API
{
	DeviceScene DeviceScene;
	IntegratorGlobals m_IntegratorGlobals;

	FrameBuffer CompositeRenderBuffer;//srgb view transformed

	FrameBuffer IrradianceRenderBuffer;
	FrameBuffer MomentsRenderBuffer;
	FrameBuffer FilteredVarianceRenderFrontBuffer;
	FrameBuffer FilteredVarianceRenderBackBuffer;
	FrameBuffer FilteredIrradianceFrontBuffer;
	FrameBuffer FilteredIrradianceBackBuffer;

	FrameBuffer AlbedoRenderBuffer;
	FrameBuffer LocalNormalsRenderBuffer;//used for normals reject
	FrameBuffer DepthRenderBuffer;//used for rejection
	FrameBuffer LocalPositionsRenderBuffer;//used for reproj & depth reject
	FrameBuffer ObjectIDRenderBuffer; //used for reproj & reject

	FrameBuffer VelocityRenderBuffer;//used for reproj

	//history
	FrameBuffer HistoryIntegratedIrradianceRenderFrontBuffer;//read
	FrameBuffer HistoryIntegratedIrradianceRenderBackBuffer;//write; Filtered output
	FrameBuffer HistoryIntegratedMomentsFrontBuffer;
	FrameBuffer HistoryIntegratedMomentsBackBuffer;
	FrameBuffer HistoryDepthRenderBuffer;
	FrameBuffer HistoryWorldNormalsRenderBuffer;

	//DEBUG
	FrameBuffer WorldNormalsRenderBuffer;//debug
	FrameBuffer PositionsRenderBuffer;//debug
	FrameBuffer ObjectIDDebugRenderBuffer;
	FrameBuffer GASDebugRenderBuffer;
	FrameBuffer HitHeatmapDebugRenderBuffer;
	FrameBuffer BboxHeatmapDebugRenderBuffer;
	FrameBuffer UVsDebugRenderBuffer;
	FrameBuffer BarycentricsDebugRenderBuffer;

	thrust::device_vector<float3>AccumulationFrameBuffer;
	thrust::device_vector<float3>VarianceAccumulationFrameBuffer;
};

Renderer::Renderer()
{
	m_CudaResourceAPI = new CudaAPI();
	m_CelestiumPTResourceAPI = new CelestiumPT_API();
	//TODO: redundant
	m_CudaResourceAPI->m_BlockGridDimensions = dim3(m_NativeRenderResolutionWidth / m_ThreadBlock_x + 1, m_NativeRenderResolutionHeight / m_ThreadBlock_y + 1);
	m_CudaResourceAPI->m_ThreadBlockDimensions = dim3(m_ThreadBlock_x, m_ThreadBlock_y);

	hipMallocManaged(&m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.DeviceGeometryAggregate, sizeof(SceneGeometry));

	m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.DeviceGeometryAggregate->SkyLight = InfiniteLight();

	m_CelestiumPTResourceAPI->DeviceScene = DeviceScene(m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.DeviceGeometryAggregate);
	m_CurrentScene = HostScene(&(m_CelestiumPTResourceAPI->DeviceScene));
}

void Renderer::resizeResolution(int width, int height)
{
	if (width == m_NativeRenderResolutionWidth && height == m_NativeRenderResolutionHeight)return;
	m_NativeRenderResolutionHeight = height;
	m_NativeRenderResolutionWidth = width;

	m_CelestiumPTResourceAPI->FilteredVarianceRenderFrontBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->CompositeRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->AlbedoRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->WorldNormalsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->LocalNormalsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->PositionsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->DepthRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryIntegratedIrradianceRenderFrontBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryIntegratedIrradianceRenderBackBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryWorldNormalsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->GASDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HitHeatmapDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->BboxHeatmapDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->UVsDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->BarycentricsDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->ObjectIDRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->VelocityRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	m_CelestiumPTResourceAPI->IrradianceRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->MomentsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->FilteredVarianceRenderBackBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->FilteredIrradianceFrontBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->FilteredIrradianceBackBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	// History Buffers
	m_CelestiumPTResourceAPI->HistoryIntegratedMomentsFrontBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryIntegratedMomentsBackBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	m_CudaResourceAPI->m_BlockGridDimensions = dim3(m_NativeRenderResolutionWidth / m_ThreadBlock_x + 1, m_NativeRenderResolutionHeight / m_ThreadBlock_y + 1);
	m_CudaResourceAPI->m_ThreadBlockDimensions = dim3(m_ThreadBlock_x, m_ThreadBlock_y);

	m_CelestiumPTResourceAPI->AccumulationFrameBuffer.resize(m_NativeRenderResolutionHeight * m_NativeRenderResolutionWidth);
	m_CelestiumPTResourceAPI->VarianceAccumulationFrameBuffer.resize(m_NativeRenderResolutionHeight * m_NativeRenderResolutionWidth);
}

static uint32_t g_frameIndex = 1;

void texCopy(const FrameBuffer& src, const FrameBuffer& dst, int t_width, int t_height) {
	glCopyImageSubData(
		src.m_RenderTargetTextureName, GL_TEXTURE_2D, 0, 0, 0, 0,
		dst.m_RenderTargetTextureName, GL_TEXTURE_2D, 0, 0, 0, 0,
		t_width, t_height, 1);
	glFinish();
	GLenum err = glGetError();
	if (err != GL_NO_ERROR) {
		printf("%s\n", glErrorString(err));
	}
}

void Renderer::renderFrame()
{
	//pre render-------------------------------------------------------------
	{
		m_CelestiumPTResourceAPI->FilteredVarianceRenderFrontBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.filtered_variance_render_front_surfobj));
		m_CelestiumPTResourceAPI->CompositeRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.composite_render_surface_object));
		m_CelestiumPTResourceAPI->AlbedoRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.albedo_render_surface_object));
		m_CelestiumPTResourceAPI->WorldNormalsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.world_normals_render_surface_object));
		m_CelestiumPTResourceAPI->LocalNormalsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.local_normals_render_surface_object));
		m_CelestiumPTResourceAPI->PositionsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.positions_render_surface_object));
		m_CelestiumPTResourceAPI->DepthRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.depth_render_surface_object));
		m_CelestiumPTResourceAPI->HistoryIntegratedIrradianceRenderFrontBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_integrated_irradiance_front_surfobj));
		m_CelestiumPTResourceAPI->HistoryIntegratedIrradianceRenderBackBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_integrated_irradiance_back_surfobj));
		m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_depth_render_surface_object));
		m_CelestiumPTResourceAPI->HistoryWorldNormalsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_world_normals_render_surface_object));
		m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.local_positions_render_surface_object));
		m_CelestiumPTResourceAPI->GASDebugRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.GAS_debug_render_surface_object));
		m_CelestiumPTResourceAPI->HitHeatmapDebugRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.heatmap_debug_render_surface_object));
		m_CelestiumPTResourceAPI->BboxHeatmapDebugRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.bbox_heatmap_debug_render_surface_object));
		m_CelestiumPTResourceAPI->UVsDebugRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.UV_debug_render_surface_object));
		m_CelestiumPTResourceAPI->BarycentricsDebugRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.bary_debug_render_surface_object));
		m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.objectID_debug_render_surface_object));
		m_CelestiumPTResourceAPI->ObjectIDRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.objectID_render_surface_object));
		m_CelestiumPTResourceAPI->VelocityRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.velocity_render_surface_object));
		m_CelestiumPTResourceAPI->IrradianceRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.current_irradiance_render_surface_object));
		m_CelestiumPTResourceAPI->MomentsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.current_moments_render_surface_object));
		m_CelestiumPTResourceAPI->FilteredVarianceRenderBackBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.filtered_variance_render_back_surfobj));
		m_CelestiumPTResourceAPI->FilteredIrradianceFrontBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.filtered_irradiance_front_render_surface_object));
		m_CelestiumPTResourceAPI->FilteredIrradianceBackBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.filtered_irradiance_back_render_surface_object));

		// History Buffers
		m_CelestiumPTResourceAPI->HistoryIntegratedMomentsFrontBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_integrated_moments_front_surfobj));
		m_CelestiumPTResourceAPI->HistoryIntegratedMomentsBackBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_integrated_moments_back_surfobj));
	}

	//prepare globals--------------------
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameIndex = g_frameIndex;
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.resolution =
		make_int2(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.accumulation_framebuffer =
		thrust::raw_pointer_cast(m_CelestiumPTResourceAPI->AccumulationFrameBuffer.data());
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.variance_accumulation_framebuffer =
		thrust::raw_pointer_cast(m_CelestiumPTResourceAPI->VarianceAccumulationFrameBuffer.data());

	//Launch RenderChain

	if (m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.use_SVGF)
	{
		renderPathTraceRaw << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);

		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		//-----------
		temporalIntegrate << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		//moments
		texCopy(m_CelestiumPTResourceAPI->HistoryIntegratedMomentsBackBuffer,
			m_CelestiumPTResourceAPI->HistoryIntegratedMomentsFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);

		//-----------

		for (int iter = 0; iter < 5; iter++) {
		
		}
		SVGFPass << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals, 1);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		texCopy(m_CelestiumPTResourceAPI->FilteredIrradianceBackBuffer,
			m_CelestiumPTResourceAPI->FilteredIrradianceFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		texCopy(m_CelestiumPTResourceAPI->FilteredVarianceRenderBackBuffer,
			m_CelestiumPTResourceAPI->FilteredVarianceRenderFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		//irradiance feedback: TODO: basically bo need for hist_int_irr back buffer
		texCopy(m_CelestiumPTResourceAPI->FilteredIrradianceBackBuffer,
			m_CelestiumPTResourceAPI->HistoryIntegratedIrradianceRenderFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		//-----------
		SVGFPass << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals, 2);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		texCopy(m_CelestiumPTResourceAPI->FilteredIrradianceBackBuffer,
			m_CelestiumPTResourceAPI->FilteredIrradianceFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		texCopy(m_CelestiumPTResourceAPI->FilteredVarianceRenderBackBuffer,
			m_CelestiumPTResourceAPI->FilteredVarianceRenderFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		//-----------
		SVGFPass << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals, 4);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		texCopy(m_CelestiumPTResourceAPI->FilteredIrradianceBackBuffer,
			m_CelestiumPTResourceAPI->FilteredIrradianceFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		texCopy(m_CelestiumPTResourceAPI->FilteredVarianceRenderBackBuffer,
			m_CelestiumPTResourceAPI->FilteredVarianceRenderFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		//-----------
		SVGFPass << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals, 8);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		texCopy(m_CelestiumPTResourceAPI->FilteredIrradianceBackBuffer,
			m_CelestiumPTResourceAPI->FilteredIrradianceFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		texCopy(m_CelestiumPTResourceAPI->FilteredVarianceRenderBackBuffer,
			m_CelestiumPTResourceAPI->FilteredVarianceRenderFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		//-----------
		composeCompositeImage << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);//Display!
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
	}
	else if (m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.temporal_accumulation)
	{
		renderPathTraceRaw << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		//---
		temporalAccumulate << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		texCopy(m_CelestiumPTResourceAPI->HistoryIntegratedMomentsBackBuffer,
			m_CelestiumPTResourceAPI->HistoryIntegratedMomentsFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		texCopy(m_CelestiumPTResourceAPI->HistoryIntegratedIrradianceRenderBackBuffer,
			m_CelestiumPTResourceAPI->HistoryIntegratedIrradianceRenderFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		//---

		composeCompositeImage << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);//Display!
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
	}
	else
	{
		renderPathTraceRaw << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		texCopy(m_CelestiumPTResourceAPI->IrradianceRenderBuffer,
			m_CelestiumPTResourceAPI->FilteredIrradianceFrontBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
		composeCompositeImage << < m_CudaResourceAPI->m_BlockGridDimensions,
			m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);//Display!
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
	}
	g_frameIndex++;

	//----
	{
		//post render cuda---------------------------------------------------------------------------------
		m_CelestiumPTResourceAPI->FilteredVarianceRenderFrontBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.filtered_variance_render_front_surfobj));
		m_CelestiumPTResourceAPI->CompositeRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.composite_render_surface_object));
		m_CelestiumPTResourceAPI->AlbedoRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.albedo_render_surface_object));
		m_CelestiumPTResourceAPI->UVsDebugRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.UV_debug_render_surface_object));
		m_CelestiumPTResourceAPI->BarycentricsDebugRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.bary_debug_render_surface_object));
		m_CelestiumPTResourceAPI->WorldNormalsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.world_normals_render_surface_object));
		m_CelestiumPTResourceAPI->LocalNormalsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.local_normals_render_surface_object));
		m_CelestiumPTResourceAPI->DepthRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.depth_render_surface_object));
		m_CelestiumPTResourceAPI->HistoryIntegratedIrradianceRenderFrontBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_integrated_irradiance_front_surfobj));
		m_CelestiumPTResourceAPI->HistoryIntegratedIrradianceRenderBackBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_integrated_irradiance_back_surfobj));
		m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_depth_render_surface_object));
		m_CelestiumPTResourceAPI->HistoryWorldNormalsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_world_normals_render_surface_object));
		m_CelestiumPTResourceAPI->PositionsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.positions_render_surface_object));
		m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.local_positions_render_surface_object));
		m_CelestiumPTResourceAPI->GASDebugRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.GAS_debug_render_surface_object));
		m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.objectID_debug_render_surface_object));
		m_CelestiumPTResourceAPI->BboxHeatmapDebugRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.bbox_heatmap_debug_render_surface_object));
		m_CelestiumPTResourceAPI->ObjectIDRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.objectID_render_surface_object));
		m_CelestiumPTResourceAPI->HitHeatmapDebugRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.heatmap_debug_render_surface_object));
		m_CelestiumPTResourceAPI->VelocityRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.velocity_render_surface_object));
		m_CelestiumPTResourceAPI->IrradianceRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.current_irradiance_render_surface_object));
		m_CelestiumPTResourceAPI->MomentsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.current_moments_render_surface_object));
		m_CelestiumPTResourceAPI->FilteredVarianceRenderBackBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.filtered_variance_render_back_surfobj));
		m_CelestiumPTResourceAPI->FilteredIrradianceFrontBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.filtered_irradiance_front_render_surface_object));
		m_CelestiumPTResourceAPI->FilteredIrradianceBackBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.filtered_irradiance_back_render_surface_object));

		// History Buffers
		m_CelestiumPTResourceAPI->HistoryIntegratedMomentsFrontBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_integrated_moments_front_surfobj));
		m_CelestiumPTResourceAPI->HistoryIntegratedMomentsBackBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_integrated_moments_back_surfobj));
	}

	//UPDATE HISTORY
	//GBUFFER blit

	//depth
	texCopy(m_CelestiumPTResourceAPI->DepthRenderBuffer,
		m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer, m_NativeRenderResolutionWidth,
		m_NativeRenderResolutionHeight);

	//normals
	texCopy(m_CelestiumPTResourceAPI->WorldNormalsRenderBuffer,
		m_CelestiumPTResourceAPI->HistoryWorldNormalsRenderBuffer, m_NativeRenderResolutionWidth,
		m_NativeRenderResolutionHeight);
}

void Renderer::clearAccumulation()
{
	if (!m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.accumulate)return;

	thrust::fill(m_CelestiumPTResourceAPI->AccumulationFrameBuffer.begin(),
		m_CelestiumPTResourceAPI->AccumulationFrameBuffer.end(), make_float3(0, 0, 0));
	thrust::fill(m_CelestiumPTResourceAPI->VarianceAccumulationFrameBuffer.begin(),
		m_CelestiumPTResourceAPI->VarianceAccumulationFrameBuffer.end(), make_float3(0, 0, 0));

	g_frameIndex = 1;
}

GLuint Renderer::getCompositeRenderTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->CompositeRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getAlbedoRenderTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->AlbedoRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getNormalsTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->WorldNormalsRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getPositionsTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->PositionsRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getLocalPositionsTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getVelocityTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->VelocityRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getGASDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->GASDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getHeatmapDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->HitHeatmapDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getBboxHeatmapDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->BboxHeatmapDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getDepthTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->DepthRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getUVsDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->UVsDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getObjectIDDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getBarycentricsDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->BarycentricsDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getAlbedoTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->AlbedoRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getIntegratedVarianceTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->FilteredVarianceRenderFrontBuffer.m_RenderTargetTextureName;
}

int Renderer::getSPP() const
{
	return g_frameIndex;
}

void Renderer::setCamera(int idx)
{
	if (idx < 0)idx = 0;
	if (idx >= m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras.size())idx = m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras.size() - 1;
	DeviceCamera* dcam = thrust::raw_pointer_cast(&m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras[idx]);
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.ActiveCamera = dcam;
	m_CurrentCamera = HostCamera(m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.ActiveCamera);
}

IntegratorSettings* Renderer::getIntegratorSettings()
{
	return &(m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG);
}

Renderer::~Renderer()
{
	hipFree(m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.DeviceGeometryAggregate);//TODO: non critical ownership issues with devicescene
	delete m_CudaResourceAPI;
	m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras.clear();
	delete m_CelestiumPTResourceAPI;
}