#include "hip/hip_runtime.h"
#include "renderer.hpp"
#include "error_check.cuh"
#include "storage.cuh"
#include "integrator.cuh"
#include "denoiser/denoiser.cuh"
#include "render_passes.cuh"
#include "maths/constants.cuh"

#include "scene_geometry.cuh"
#include "device_scene.cuh"
#include "device_camera.cuh"
//#include "Triangle.cuh"

#include <cuda_gl_interop.h>
#include <iostream>
/*
* Mostly adapter and inter conversion work
*/

constexpr int SVGF_MAX_ITERATIONS = 4;
constexpr int ASVGF_MAX_ITERATIONS = 5;

class FrameBuffer {
public:

	FrameBuffer() = default;

	void initialize(int width, int height)
	{
		//GL texture configure
		glGenTextures(1, &m_RenderTargetTextureName);
		glBindTexture(GL_TEXTURE_2D, m_RenderTargetTextureName);

		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
		//TODO: make a switchable frame filtering mode
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA, GL_FLOAT, NULL);

		glBindTexture(GL_TEXTURE_2D, 0);

		hipGraphicsGLRegisterImage(&m_RenderTargetTextureCudaResource, m_RenderTargetTextureName, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
	};

	~FrameBuffer() {
		glDeleteTextures(1, &m_RenderTargetTextureName);
	}

	void resizeResolution(int width, int height) {
		if (m_RenderTargetTextureName == NULL) {
			initialize(width, height); return;
		}
		// unregister
		hipGraphicsUnregisterResource(m_RenderTargetTextureCudaResource);
		// resize
		glBindTexture(GL_TEXTURE_2D, m_RenderTargetTextureName);
		{
			glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA, GL_FLOAT, NULL);
		}
		glBindTexture(GL_TEXTURE_2D, 0);
		// register back
		hipGraphicsGLRegisterImage(&m_RenderTargetTextureCudaResource, m_RenderTargetTextureName, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
	};

	void beginRender(hipSurfaceObject_t* surf_obj) {
		hipGraphicsMapResources(1, &m_RenderTargetTextureCudaResource);

		hipArray_t render_target_texture_sub_resource_array;
		hipGraphicsSubResourceGetMappedArray(&render_target_texture_sub_resource_array, m_RenderTargetTextureCudaResource,
			0, 0);
		hipResourceDesc render_target_texture_resource_descriptor;
		{
			render_target_texture_resource_descriptor.resType = hipResourceTypeArray;
			render_target_texture_resource_descriptor.res.array.array = render_target_texture_sub_resource_array;
		}

		//prepare globals--------------------
		hipCreateSurfaceObject(surf_obj, &render_target_texture_resource_descriptor);
	}

	void endRender(hipSurfaceObject_t* surf_obj)
	{
		hipDestroySurfaceObject(*surf_obj);
		hipGraphicsUnmapResources(1, &m_RenderTargetTextureCudaResource);
		hipStreamSynchronize(0);
	}

	GLuint m_RenderTargetTextureName = NULL;
	hipGraphicsResource_t m_RenderTargetTextureCudaResource;
};

struct CudaAPI
{
	dim3 m_BlockGridDimensions;
	dim3 m_ThreadBlockDimensions;
};

struct CelestiumPT_API
{
	DeviceScene DeviceScene;
	IntegratorGlobals m_IntegratorGlobals;

	//post
	FrameBuffer CompositeRenderBuffer;//srgb view transformed

	//raw
	FrameBuffer RawIrradianceRenderBuffer;

	//SVGF
	FrameBuffer SVGFFilteredIrradianceFrontBuffer;
	FrameBuffer SVGFFilteredIrradianceBackBuffer;
	FrameBuffer SVGFFilteredVarianceRenderFrontBuffer;
	FrameBuffer SVGFFilteredVarianceRenderBackBuffer;

	//GBUFFER
	FrameBuffer AlbedoRenderBuffer;
	FrameBuffer LocalNormalsRenderBuffer;//used for normals reject
	FrameBuffer LocalPositionsRenderBuffer;//used for reproj & depth reject
	FrameBuffer WorldNormalsRenderBuffer;
	FrameBuffer WorldPositionsRenderBuffer;

	FrameBuffer DepthRenderBuffer;//used for rejection

	FrameBuffer UVsRenderBuffer;
	FrameBuffer BarycentricsRenderBuffer;
	FrameBuffer ObjectIDRenderBuffer; //used for reproj & reject
	FrameBuffer TriangleIDRenderBuffer;
	FrameBuffer VelocityRenderBuffer;//used for reproj

	//adaptive filter
	FrameBuffer SeedsBuffer;

	FrameBuffer HistoryShadingBuffer;
	FrameBuffer HistorySeedsBuffer;

	FrameBuffer HistoryObjectIDBuffer;
	FrameBuffer HistoryTriangleIDBuffer;

	FrameBuffer HistoryLocalPositionsBuffer;
	FrameBuffer HistoryWorldPositionsBuffer;
	FrameBuffer HistoryLocalNormalsBuffer;

	FrameBuffer HistoryUVsBuffer;
	FrameBuffer HistoryBarycentricsBuffer;

	FrameBuffer SparseGradientBuffer;
	FrameBuffer DenseGradientFrontBuffer;
	FrameBuffer DenseGradientBackBuffer;

	//temporal filter
	FrameBuffer IntegratedMomentsFrontBuffer;
	FrameBuffer IntegratedMomentsBackBuffer;
	FrameBuffer IntegratedIrradianceRenderFrontBuffer;//read
	FrameBuffer IntegratedIrradianceRenderBackBuffer;//write; Filtered output

	//history
	FrameBuffer HistoryDepthRenderBuffer;
	FrameBuffer HistoryWorldNormalsRenderBuffer;

	//debug
	FrameBuffer MiscDebugViewBuffer;
	FrameBuffer ObjectIDDebugRenderBuffer;
	FrameBuffer GASDebugRenderBuffer;
	FrameBuffer HitHeatmapDebugRenderBuffer;
	FrameBuffer BboxHeatmapDebugRenderBuffer;

	//static accum
	thrust::device_vector<float3>AccumulationFrameBuffer;
	thrust::device_vector<float3>VarianceAccumulationFrameBuffer;
};

Renderer::Renderer()
{
	m_CudaResourceAPI = new CudaAPI();
	m_CelestiumPTResourceAPI = new CelestiumPT_API();
	//TODO: redundant
	m_CudaResourceAPI->m_BlockGridDimensions = dim3(m_NativeRenderResolutionWidth / m_ThreadBlock_x + 1, m_NativeRenderResolutionHeight / m_ThreadBlock_y + 1);
	m_CudaResourceAPI->m_ThreadBlockDimensions = dim3(m_ThreadBlock_x, m_ThreadBlock_y);

	hipMallocManaged(&m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.DeviceGeometryAggregate, sizeof(SceneGeometry));

	m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.DeviceGeometryAggregate->SkyLight = InfiniteLight();

	m_CelestiumPTResourceAPI->DeviceScene = DeviceScene(m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.DeviceGeometryAggregate);
	m_CurrentScene = HostScene(&(m_CelestiumPTResourceAPI->DeviceScene));
}

void Renderer::resizeResolution(int width, int height)
{
	if (width == m_NativeRenderResolutionWidth && height == m_NativeRenderResolutionHeight)return;
	m_NativeRenderResolutionHeight = height;
	m_NativeRenderResolutionWidth = width;

	//post
	m_CelestiumPTResourceAPI->CompositeRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	//raw
	m_CelestiumPTResourceAPI->RawIrradianceRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	//svgf
	m_CelestiumPTResourceAPI->SVGFFilteredVarianceRenderFrontBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->SVGFFilteredVarianceRenderBackBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->SVGFFilteredIrradianceFrontBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->SVGFFilteredIrradianceBackBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	//gbuffer
	m_CelestiumPTResourceAPI->AlbedoRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	m_CelestiumPTResourceAPI->WorldNormalsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->LocalNormalsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->WorldPositionsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	m_CelestiumPTResourceAPI->DepthRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	m_CelestiumPTResourceAPI->UVsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->BarycentricsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->ObjectIDRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->TriangleIDRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->VelocityRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	//temporal filter
	m_CelestiumPTResourceAPI->IntegratedIrradianceRenderFrontBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->IntegratedIrradianceRenderBackBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->IntegratedMomentsFrontBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->IntegratedMomentsBackBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	//asvgf
	m_CelestiumPTResourceAPI->SparseGradientBuffer.resizeResolution(
		(m_NativeRenderResolutionWidth + ASVGF_STRATUM_SIZE - 1) / ASVGF_STRATUM_SIZE,
		(m_NativeRenderResolutionHeight + ASVGF_STRATUM_SIZE - 1) / ASVGF_STRATUM_SIZE);
	m_CelestiumPTResourceAPI->DenseGradientFrontBuffer.resizeResolution(
		(m_NativeRenderResolutionWidth + ASVGF_STRATUM_SIZE - 1) / ASVGF_STRATUM_SIZE,
		(m_NativeRenderResolutionHeight + ASVGF_STRATUM_SIZE - 1) / ASVGF_STRATUM_SIZE);
	m_CelestiumPTResourceAPI->DenseGradientBackBuffer.resizeResolution(
		(m_NativeRenderResolutionWidth + ASVGF_STRATUM_SIZE - 1) / ASVGF_STRATUM_SIZE,
		(m_NativeRenderResolutionHeight + ASVGF_STRATUM_SIZE - 1) / ASVGF_STRATUM_SIZE);

	m_CelestiumPTResourceAPI->SeedsBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryShadingBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistorySeedsBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	m_CelestiumPTResourceAPI->HistoryLocalPositionsBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryWorldPositionsBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryLocalNormalsBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryUVsBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryBarycentricsBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryTriangleIDBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryObjectIDBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	//debugviews
	m_CelestiumPTResourceAPI->GASDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HitHeatmapDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->BboxHeatmapDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->MiscDebugViewBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	// History Buffers
	m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->HistoryWorldNormalsRenderBuffer.resizeResolution(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);

	//static accum
	m_CelestiumPTResourceAPI->AccumulationFrameBuffer.resize(m_NativeRenderResolutionHeight * m_NativeRenderResolutionWidth);
	m_CelestiumPTResourceAPI->VarianceAccumulationFrameBuffer.resize(m_NativeRenderResolutionHeight * m_NativeRenderResolutionWidth);

	//=============================
	m_CudaResourceAPI->m_BlockGridDimensions = dim3(m_NativeRenderResolutionWidth / m_ThreadBlock_x + 1, m_NativeRenderResolutionHeight / m_ThreadBlock_y + 1);
	m_CudaResourceAPI->m_ThreadBlockDimensions = dim3(m_ThreadBlock_x, m_ThreadBlock_y);
}

static uint32_t g_frameIndex = 1;

void texCopy(const FrameBuffer& src, const FrameBuffer& dst, int t_width, int t_height) {
	glCopyImageSubData(
		src.m_RenderTargetTextureName, GL_TEXTURE_2D, 0, 0, 0, 0,
		dst.m_RenderTargetTextureName, GL_TEXTURE_2D, 0, 0, 0, 0,
		t_width, t_height, 1);
	glFinish();
	GLenum err = glGetError();
	if (err != GL_NO_ERROR) {
		printf("[TEXCOPY]: %s\n", glErrorString(err));
	}
}

void Renderer::renderFrame()
{
	//pre render-------------------------------------------------------------
	{
		// Post ---------------------------------
		m_CelestiumPTResourceAPI->CompositeRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.composite_surfobject));

		// Raw Buffers ---------------------------------
		m_CelestiumPTResourceAPI->RawIrradianceRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.raw_irradiance_surfobject));

		// SVGF Buffers ---------------------------------
		m_CelestiumPTResourceAPI->SVGFFilteredVarianceRenderFrontBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.svgf_filtered_variance_front_surfobject));
		m_CelestiumPTResourceAPI->SVGFFilteredVarianceRenderBackBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.svgf_filtered_variance_back_surfobject));
		m_CelestiumPTResourceAPI->SVGFFilteredIrradianceFrontBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.svgf_filtered_irradiance_front_surfobject));
		m_CelestiumPTResourceAPI->SVGFFilteredIrradianceBackBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.svgf_filtered_irradiance_back_surfobject));

		// Albedo Buffer ---------------------------------
		m_CelestiumPTResourceAPI->AlbedoRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.albedo_surfobject));

		// G-Buffer ---------------------------------
		m_CelestiumPTResourceAPI->LocalNormalsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.local_normals_surfobject));
		m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.local_positions_surfobject));
		m_CelestiumPTResourceAPI->WorldNormalsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.world_normals_surfobject));
		m_CelestiumPTResourceAPI->WorldPositionsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.world_positions_surfobject));
		m_CelestiumPTResourceAPI->DepthRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.depth_surfobject));
		m_CelestiumPTResourceAPI->UVsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.UVs_surfobject));
		m_CelestiumPTResourceAPI->BarycentricsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.bary_surfobject));
		m_CelestiumPTResourceAPI->ObjectIDRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.objectID_surfobject));
		m_CelestiumPTResourceAPI->TriangleIDRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.triangleID_surfobject));
		m_CelestiumPTResourceAPI->VelocityRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.velocity_surfobject));

		// Temporal Accumulation Buffers ---------------------------------
		m_CelestiumPTResourceAPI->IntegratedIrradianceRenderFrontBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.integrated_irradiance_front_surfobject));
		m_CelestiumPTResourceAPI->IntegratedIrradianceRenderBackBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.integrated_irradiance_back_surfobject));
		m_CelestiumPTResourceAPI->IntegratedMomentsFrontBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.integrated_moments_front_surfobject));
		m_CelestiumPTResourceAPI->IntegratedMomentsBackBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.integrated_moments_back_surfobject));

		// ASVGF -------------------------------
		m_CelestiumPTResourceAPI->SparseGradientBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.asvgf_sparse_gradient_surfobject));
		m_CelestiumPTResourceAPI->DenseGradientFrontBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.asvgf_dense_gradient_front_surfobject));
		m_CelestiumPTResourceAPI->DenseGradientBackBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.asvgf_dense_gradient_back_surfobject));

		m_CelestiumPTResourceAPI->SeedsBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.seeds_surfobject));
		m_CelestiumPTResourceAPI->HistoryShadingBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_shading_surfobject));
		m_CelestiumPTResourceAPI->HistorySeedsBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_seeds_surfobject));

		m_CelestiumPTResourceAPI->HistoryLocalPositionsBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_local_positions_surfobject));
		m_CelestiumPTResourceAPI->HistoryWorldPositionsBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_world_positions_surfobject));
		m_CelestiumPTResourceAPI->HistoryLocalNormalsBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_local_normals_surfobject));
		m_CelestiumPTResourceAPI->HistoryUVsBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_UVs_surfobject));
		m_CelestiumPTResourceAPI->HistoryBarycentricsBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_bary_surfobject));
		m_CelestiumPTResourceAPI->HistoryTriangleIDBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_triangleID_surfobject));
		m_CelestiumPTResourceAPI->HistoryObjectIDBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_objectID_surfobject));

		// History Buffers ---------------------------------
		m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_depth_surfobject));
		m_CelestiumPTResourceAPI->HistoryWorldNormalsRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_world_normals_surfobject));

		// Debug Buffers ---------------------------------
		m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.debugview_objectID_surfobject));
		m_CelestiumPTResourceAPI->MiscDebugViewBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.debugview_misc_surfobject));

		m_CelestiumPTResourceAPI->GASDebugRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.debugview_GAS_overlap_surfobject));
		m_CelestiumPTResourceAPI->HitHeatmapDebugRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.debugview_tri_test_heatmap_surfobject));
		m_CelestiumPTResourceAPI->BboxHeatmapDebugRenderBuffer.beginRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.debugview_bbox_test_heatmap_surfobject));
	}

	//prepare globals--------------------
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameIndex = g_frameIndex;
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.resolution =
		make_int2(m_NativeRenderResolutionWidth, m_NativeRenderResolutionHeight);
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.accumulation_framebuffer =
		thrust::raw_pointer_cast(m_CelestiumPTResourceAPI->AccumulationFrameBuffer.data());
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.variance_accumulation_framebuffer =
		thrust::raw_pointer_cast(m_CelestiumPTResourceAPI->VarianceAccumulationFrameBuffer.data());

	//Launch RenderChain
	{
		//Compute primary visbility=========================================
		{
			computePrimaryVisibility << < m_CudaResourceAPI->m_BlockGridDimensions,
				m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);
			//sync
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());
		}

		//Merge shading and surface samples==========================================
		if (m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.adaptive_temporal_filter_enabled)
		{
			mergeSamples << < m_CudaResourceAPI->m_BlockGridDimensions,
				m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);
			//sync
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());
		}

		//Compute pathtrace samples==========================================
		{
			tracePathSample << < m_CudaResourceAPI->m_BlockGridDimensions,
				m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);
			//sync
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());
			//potential skips----------------
			if (!m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.svgf_enabled &&
				!m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.temporal_filter_enabled)
				texCopy(m_CelestiumPTResourceAPI->RawIrradianceRenderBuffer,
					m_CelestiumPTResourceAPI->SVGFFilteredIrradianceFrontBuffer, m_NativeRenderResolutionWidth,
					m_NativeRenderResolutionHeight);
		}

		//Compute gradient samples==========================================
		if (m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.adaptive_temporal_filter_enabled)
		{
			createGradientSamples << < m_CudaResourceAPI->m_BlockGridDimensions,
				m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);
			//sync
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());
			texCopy(m_CelestiumPTResourceAPI->SparseGradientBuffer,
				m_CelestiumPTResourceAPI->DenseGradientFrontBuffer,
				(m_NativeRenderResolutionWidth + ASVGF_STRATUM_SIZE - 1) / ASVGF_STRATUM_SIZE,
				(m_NativeRenderResolutionHeight + ASVGF_STRATUM_SIZE - 1) / ASVGF_STRATUM_SIZE);
		}

		//Gradient atrous===================================================
		if (m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.adaptive_temporal_filter_enabled)
		{
			for (int iter = 0; iter < ASVGF_MAX_ITERATIONS; iter++)
			{
				int stepsize = pow(2, iter);

				atrousGradient << < m_CudaResourceAPI->m_BlockGridDimensions,
					m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals, stepsize);
				//sync-------
				checkCudaErrors(hipGetLastError());
				checkCudaErrors(hipDeviceSynchronize());

				//working buffers swap----------
				texCopy(m_CelestiumPTResourceAPI->DenseGradientBackBuffer,
					m_CelestiumPTResourceAPI->DenseGradientFrontBuffer,
					(m_NativeRenderResolutionWidth + ASVGF_STRATUM_SIZE - 1) / ASVGF_STRATUM_SIZE,
					(m_NativeRenderResolutionHeight + ASVGF_STRATUM_SIZE - 1) / ASVGF_STRATUM_SIZE);
			}
		}

		//Temporal accumulation===========================================
		if (m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.svgf_enabled ||
			m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.temporal_filter_enabled)
		{
			temporalAccumulate << < m_CudaResourceAPI->m_BlockGridDimensions,
				m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);
			//sync
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());
			//moments swap
			texCopy(m_CelestiumPTResourceAPI->IntegratedMomentsBackBuffer,
				m_CelestiumPTResourceAPI->IntegratedMomentsFrontBuffer, m_NativeRenderResolutionWidth,
				m_NativeRenderResolutionHeight);
			//potential irradiance feedback
			if (!m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.svgf_enabled) {
				texCopy(m_CelestiumPTResourceAPI->SVGFFilteredIrradianceFrontBuffer,
					m_CelestiumPTResourceAPI->IntegratedIrradianceRenderFrontBuffer, m_NativeRenderResolutionWidth,
					m_NativeRenderResolutionHeight);
			}

			//Variance Estimation=================================================
			estimateVariance << < m_CudaResourceAPI->m_BlockGridDimensions,
				m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);
			//sync
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());
		}

		//DEBUG
		//texCopy(m_CelestiumPTResourceAPI->RawIrradianceRenderBuffer,
		//	m_CelestiumPTResourceAPI->MiscDebugViewBuffer, m_NativeRenderResolutionWidth,
		//	m_NativeRenderResolutionHeight);

		//SVGF atrous====================================
		if (m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.svgf_enabled)
		{
			for (int iter = 0; iter < SVGF_MAX_ITERATIONS; iter++)
			{
				int stepsize = pow(2, iter);

				atrousSVGF << < m_CudaResourceAPI->m_BlockGridDimensions,
					m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals, stepsize);
				//sync-------
				checkCudaErrors(hipGetLastError());
				checkCudaErrors(hipDeviceSynchronize());

				//irradiance feedback after 1st iter: TODO: basically no need for hist_int_irr back buffer
				if (iter == 0) {
					texCopy(m_CelestiumPTResourceAPI->SVGFFilteredIrradianceBackBuffer,
						m_CelestiumPTResourceAPI->IntegratedIrradianceRenderFrontBuffer, m_NativeRenderResolutionWidth,
						m_NativeRenderResolutionHeight);
				}

				//working buffers swap----------
				texCopy(m_CelestiumPTResourceAPI->SVGFFilteredIrradianceBackBuffer,
					m_CelestiumPTResourceAPI->SVGFFilteredIrradianceFrontBuffer, m_NativeRenderResolutionWidth,
					m_NativeRenderResolutionHeight);
				texCopy(m_CelestiumPTResourceAPI->SVGFFilteredVarianceRenderBackBuffer,
					m_CelestiumPTResourceAPI->SVGFFilteredVarianceRenderFrontBuffer, m_NativeRenderResolutionWidth,
					m_NativeRenderResolutionHeight);
			}
		}

		//Compose=====================
		{
			composeCompositeImage << < m_CudaResourceAPI->m_BlockGridDimensions,
				m_CudaResourceAPI->m_ThreadBlockDimensions >> > (m_CelestiumPTResourceAPI->m_IntegratorGlobals);
			//sync
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());
		}
	}

	g_frameIndex++;

	//post render cuda---------------------------------------------------------------------------------
	{
		// Post ---------------------------------
		m_CelestiumPTResourceAPI->CompositeRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.composite_surfobject));

		// Raw Buffers ---------------------------------
		m_CelestiumPTResourceAPI->RawIrradianceRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.raw_irradiance_surfobject));

		// SVGF Buffers ---------------------------------
		m_CelestiumPTResourceAPI->SVGFFilteredVarianceRenderFrontBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.svgf_filtered_variance_front_surfobject));
		m_CelestiumPTResourceAPI->SVGFFilteredVarianceRenderBackBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.svgf_filtered_variance_back_surfobject));
		m_CelestiumPTResourceAPI->SVGFFilteredIrradianceFrontBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.svgf_filtered_irradiance_front_surfobject));
		m_CelestiumPTResourceAPI->SVGFFilteredIrradianceBackBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.svgf_filtered_irradiance_back_surfobject));

		// Albedo Buffer ---------------------------------
		m_CelestiumPTResourceAPI->AlbedoRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.albedo_surfobject));

		// G-Buffer ---------------------------------
		m_CelestiumPTResourceAPI->LocalNormalsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.local_normals_surfobject));
		m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.local_positions_surfobject));
		m_CelestiumPTResourceAPI->WorldNormalsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.world_normals_surfobject));
		m_CelestiumPTResourceAPI->WorldPositionsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.world_positions_surfobject));
		m_CelestiumPTResourceAPI->DepthRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.depth_surfobject));
		m_CelestiumPTResourceAPI->UVsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.UVs_surfobject));
		m_CelestiumPTResourceAPI->BarycentricsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.bary_surfobject));
		m_CelestiumPTResourceAPI->ObjectIDRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.objectID_surfobject));
		m_CelestiumPTResourceAPI->TriangleIDRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.triangleID_surfobject));
		m_CelestiumPTResourceAPI->VelocityRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.velocity_surfobject));

		// Temporal Accumulation Buffers ---------------------------------
		m_CelestiumPTResourceAPI->IntegratedIrradianceRenderFrontBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.integrated_irradiance_front_surfobject));
		m_CelestiumPTResourceAPI->IntegratedIrradianceRenderBackBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.integrated_irradiance_back_surfobject));
		m_CelestiumPTResourceAPI->IntegratedMomentsFrontBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.integrated_moments_front_surfobject));
		m_CelestiumPTResourceAPI->IntegratedMomentsBackBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.integrated_moments_back_surfobject));

		//ASVGF---------------------------------------------
		m_CelestiumPTResourceAPI->SparseGradientBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.asvgf_sparse_gradient_surfobject));
		m_CelestiumPTResourceAPI->DenseGradientFrontBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.asvgf_dense_gradient_front_surfobject));
		m_CelestiumPTResourceAPI->DenseGradientBackBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.asvgf_dense_gradient_back_surfobject));

		m_CelestiumPTResourceAPI->SeedsBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.seeds_surfobject));
		m_CelestiumPTResourceAPI->HistoryShadingBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_shading_surfobject));
		m_CelestiumPTResourceAPI->HistorySeedsBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_seeds_surfobject));

		m_CelestiumPTResourceAPI->HistoryLocalPositionsBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_local_positions_surfobject));
		m_CelestiumPTResourceAPI->HistoryWorldPositionsBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_world_positions_surfobject));
		m_CelestiumPTResourceAPI->HistoryLocalNormalsBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_local_normals_surfobject));
		m_CelestiumPTResourceAPI->HistoryUVsBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_UVs_surfobject));
		m_CelestiumPTResourceAPI->HistoryBarycentricsBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_bary_surfobject));
		m_CelestiumPTResourceAPI->HistoryTriangleIDBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_triangleID_surfobject));
		m_CelestiumPTResourceAPI->HistoryObjectIDBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_objectID_surfobject));

		// History Buffers ---------------------------------
		m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_depth_surfobject));
		m_CelestiumPTResourceAPI->HistoryWorldNormalsRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.history_world_normals_surfobject));

		// Debug Buffers ---------------------------------
		m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.debugview_objectID_surfobject));
		m_CelestiumPTResourceAPI->MiscDebugViewBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.debugview_misc_surfobject));
		m_CelestiumPTResourceAPI->GASDebugRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.debugview_GAS_overlap_surfobject));
		m_CelestiumPTResourceAPI->HitHeatmapDebugRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.debugview_tri_test_heatmap_surfobject));
		m_CelestiumPTResourceAPI->BboxHeatmapDebugRenderBuffer.endRender(
			&(m_CelestiumPTResourceAPI->m_IntegratorGlobals.FrameBuffer.debugview_bbox_test_heatmap_surfobject));
	}

	//UPDATE HISTORY
	{
		//seeds
		texCopy(m_CelestiumPTResourceAPI->SeedsBuffer,
			m_CelestiumPTResourceAPI->HistorySeedsBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);

		//shading
		texCopy(m_CelestiumPTResourceAPI->RawIrradianceRenderBuffer,
			m_CelestiumPTResourceAPI->HistoryShadingBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);

		//w_normals
		texCopy(m_CelestiumPTResourceAPI->WorldNormalsRenderBuffer,
			m_CelestiumPTResourceAPI->HistoryWorldNormalsRenderBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);

		//w_pos
		texCopy(m_CelestiumPTResourceAPI->WorldPositionsRenderBuffer,
			m_CelestiumPTResourceAPI->HistoryWorldPositionsBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);

		//l_nrm
		texCopy(m_CelestiumPTResourceAPI->LocalNormalsRenderBuffer,
			m_CelestiumPTResourceAPI->HistoryLocalNormalsBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);

		//l_pos
		texCopy(m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer,
			m_CelestiumPTResourceAPI->HistoryLocalPositionsBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);

		//depth
		texCopy(m_CelestiumPTResourceAPI->DepthRenderBuffer,
			m_CelestiumPTResourceAPI->HistoryDepthRenderBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);

		//triangle ID
		texCopy(m_CelestiumPTResourceAPI->TriangleIDRenderBuffer,
			m_CelestiumPTResourceAPI->HistoryTriangleIDBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);

		//object ID
		texCopy(m_CelestiumPTResourceAPI->ObjectIDRenderBuffer,
			m_CelestiumPTResourceAPI->HistoryObjectIDBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);

		//uvs
		texCopy(m_CelestiumPTResourceAPI->UVsRenderBuffer,
			m_CelestiumPTResourceAPI->HistoryUVsBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);

		//bary
		texCopy(m_CelestiumPTResourceAPI->BarycentricsRenderBuffer,
			m_CelestiumPTResourceAPI->HistoryBarycentricsBuffer, m_NativeRenderResolutionWidth,
			m_NativeRenderResolutionHeight);
	}
}

void Renderer::clearAccumulation()
{
	if (!m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG.accumulate)return;

	thrust::fill(m_CelestiumPTResourceAPI->AccumulationFrameBuffer.begin(),
		m_CelestiumPTResourceAPI->AccumulationFrameBuffer.end(), make_float3(0, 0, 0));
	thrust::fill(m_CelestiumPTResourceAPI->VarianceAccumulationFrameBuffer.begin(),
		m_CelestiumPTResourceAPI->VarianceAccumulationFrameBuffer.end(), make_float3(0, 0, 0));

	g_frameIndex = 1;
}

GLuint Renderer::getCompositeRenderTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->CompositeRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getAlbedoRenderTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->AlbedoRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getNormalsTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->WorldNormalsRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getPositionsTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->WorldPositionsRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getLocalPositionsTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->LocalPositionsRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getVelocityTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->VelocityRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getGASDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->GASDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getHeatmapDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->HitHeatmapDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getBboxHeatmapDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->BboxHeatmapDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getDepthTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->DepthRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getUVsDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->UVsRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getObjectIDDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->ObjectIDDebugRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getBarycentricsDebugTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->BarycentricsRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getAlbedoTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->AlbedoRenderBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getIntegratedVarianceTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->SVGFFilteredVarianceRenderFrontBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getSparseGradientTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->SparseGradientBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getDenseGradientTargetTextureName() const
{
	return m_CelestiumPTResourceAPI->DenseGradientFrontBuffer.m_RenderTargetTextureName;
}

GLuint Renderer::getMiscDebugTextureName() const
{
	return m_CelestiumPTResourceAPI->MiscDebugViewBuffer.m_RenderTargetTextureName;
}

int Renderer::getSPP() const
{
	return g_frameIndex;
}

void Renderer::setCamera(int idx)
{
	if (idx < 0)idx = 0;
	if (idx >= m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras.size())idx = m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras.size() - 1;
	DeviceCamera* dcam = thrust::raw_pointer_cast(&m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras[idx]);
	m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.ActiveCamera = dcam;
	m_CurrentCamera = HostCamera(m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.ActiveCamera);
}

IntegratorSettings* Renderer::getIntegratorSettings()
{
	return &(m_CelestiumPTResourceAPI->m_IntegratorGlobals.IntegratorCFG);
}

Renderer::~Renderer()
{
	hipFree(m_CelestiumPTResourceAPI->m_IntegratorGlobals.SceneDescriptor.DeviceGeometryAggregate);//TODO: non critical ownership issues with devicescene
	delete m_CudaResourceAPI;
	m_CelestiumPTResourceAPI->DeviceScene.DeviceCameras.clear();
	delete m_CelestiumPTResourceAPI;
}