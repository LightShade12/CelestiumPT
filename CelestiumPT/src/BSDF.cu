#include "hip/hip_runtime.h"
#include "BSDF.cuh"
#include "DeviceMaterial.cuh"
#include "samplers.cuh"

BSDF::BSDF(const Mat3& tangent_matrix, const DeviceMaterial& material)
{
	tangentMatrix = tangent_matrix;
	albedo_factor = material.albedo_color_factor;
}

__device__ RGBSpectrum BSDF::f(float3 r_wo, float3 r_wi) const
{
	//float3 wo = tangentMatrix.inverse() * r_wo;
	float3 wi = tangentMatrix.inverse() * r_wi;
	float3 wo = r_wo;
	//float3 wi = r_wi;
	return fOpaqueDielectric(wo, wi);
}

__device__ float BSDF::pdf(float3 r_wo, float3 r_wi) const
{
	float3 wi = tangentMatrix.inverse() * r_wi;
	float3 wo = r_wo;
	return pdfOpaqueDielectric(wo, wi);
}

__device__ BSDFSample BSDF::sampleBSDF(float3 r_wo, float2 u2) const
{
	float3 wo = tangentMatrix.inverse() * r_wo;
	BSDFSample bs = sampleOpaqueDielectric(wo, u2);
	bs.wi = tangentMatrix * bs.wi;
	return bs;
}

__device__ BSDFSample BSDF::sampleOpaqueDielectric(float3 wo, float2 u2) const
{
	float3 wi = Samplers::sampleCosineWeightedHemisphere(u2);
	RGBSpectrum f = fOpaqueDielectric(wo, wi);
	float pdf = pdfOpaqueDielectric(wo, wi);
	return BSDFSample(f, wi, pdf);
}

__device__ RGBSpectrum BSDF::fOpaqueDielectric(float3 wo, float3 wi) const
{
	//return (albedo_factor / PI);
	return (RGBSpectrum(0.8) / PI);
}

__device__ float BSDF::pdfOpaqueDielectric(float3 wo, float3 wi) const
{
	return AbsDot({ 0,0,1 }, wi) / PI;
}