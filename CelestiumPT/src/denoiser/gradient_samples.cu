#include "hip/hip_runtime.h"
#include "denoiser.cuh"
#include "cuda_utility.cuh"
#include "maths/constants.cuh"
#include "maths/linear_algebra.cuh"

__global__ void createGradientSamples(const IntegratorGlobals t_globals)
{
	//setup threads
	int thread_pixel_coord_x = threadIdx.x + blockIdx.x * blockDim.x;
	int thread_pixel_coord_y = threadIdx.y + blockIdx.y * blockDim.y;
	int2 current_pix = make_int2(thread_pixel_coord_x, thread_pixel_coord_y);

	int2 frame_res = t_globals.FrameBuffer.resolution;
	float2 screen_uv = { (float)current_pix.x / (float)frame_res.x, (float)current_pix.y / (float)frame_res.y };

	if ((current_pix.x >= frame_res.x / ASVGF_STRATUM_SIZE) || (current_pix.y >= frame_res.y / ASVGF_STRATUM_SIZE)) return;
	//=========================================================

	int2 grad_pix = current_pix;

	float4 gradFB = texReadNearest(t_globals.FrameBuffer.asvgf_gradient_sample_surfobject, grad_pix);

	int2 pos_in_stratum = unpackStratumPos(gradFB.x);
	int2 sampling_pix = grad_pix * ASVGF_STRATUM_SIZE + pos_in_stratum;
	sampling_pix = clamp(sampling_pix, { 0,0 }, frame_res - 1);

	int current_objID = texReadNearest(t_globals.FrameBuffer.objectID_surfobject, sampling_pix).x;
	float4 seed = texReadNearest(t_globals.FrameBuffer.seeds_surfobject, sampling_pix);

	//void sample/ miss/ sky; invalid grad sample
	if (current_objID < 0 || seed.w < 0)
	{
		texWrite(make_float4(0, 0, 0, 0),
			t_globals.FrameBuffer.asvgf_sparse_gradient_surfobject, grad_pix);
		return;
	}

	//init luminance estimates

	float stratum_luminance_estimate = 0;
	float stratum_variance_estimate = 0;

	float s = 0, s2 = 0;
	constexpr int stratum_radius = ASVGF_STRATUM_SIZE / 2.f;
	int samples = 0;

	//stratum scan
	for (int y = -stratum_radius; y <= stratum_radius; y++) {
		for (int x = -stratum_radius; x <= stratum_radius; x++)
		{
			int2 offset = make_int2(x, y);
			int2 tap_pix = sampling_pix + offset;
			tap_pix = clamp(tap_pix, make_int2(0, 0), (frame_res - 1));

			//at this point history contains current frame shading
			float3 color = make_float3(texReadNearest(t_globals.FrameBuffer.raw_irradiance_surfobject, tap_pix));
			float lum = getLuminance(RGBSpectrum(color));

			//TODO: include weight factor for objectID check
			stratum_luminance_estimate += lum;
			s += lum;
			s2 += Sqr(lum);

			samples++;
		}
	}

	stratum_luminance_estimate /= samples;
	stratum_variance_estimate = fabsf((s2 / samples) - Sqr(s / samples));

	//full-res samples
	float4 current_shading = texReadNearest(t_globals.FrameBuffer.raw_irradiance_surfobject, sampling_pix);
	float4 prev_shading = texReadNearest(t_globals.FrameBuffer.history_shading_surfobject, sampling_pix);

	float c_lum = getLuminance(RGBSpectrum(current_shading));
	float p_lum = getLuminance(RGBSpectrum(prev_shading));

	float delta = c_lum - p_lum;
	float normalization_factor = fmaxf(c_lum, p_lum);

	//for debugging
	float3 delta_col = make_float3((delta < 0) ? fabsf(delta) : 0,
		(delta > 0) ? fabsf(delta) : 0,
		0);
	//delta_col = make_float3(fabsf(delta));

	texWrite(make_float4(delta_col, normalization_factor),
		t_globals.FrameBuffer.asvgf_sparse_gradient_surfobject, grad_pix);
	texWrite(make_float4(make_float3(stratum_luminance_estimate), 1),
		t_globals.FrameBuffer.asvgf_luminance_front_surfobject, grad_pix);
	texWrite(make_float4(make_float3(stratum_variance_estimate), 1),
		t_globals.FrameBuffer.asvgf_variance_front_surfobject, grad_pix);
}